/**********************************************************************
 * DESCRIPTION:
 *   Serial Concurrent Wave Equation - C Version
 *   This program implements the concurrent wave equation
 *********************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265

#define BLOCK_SIZE 32
void check_param(void);
void printfinal (void);

int nsteps,                 	/* number of time steps */
    tpoints, 	     		      /* total points along string */
    rcode;                  	/* generic return code */
float  values[MAXPOINTS+2], 	/* values at time t */
       oldval[MAXPOINTS+2], 	/* values at time (t-dt) */
       newval[MAXPOINTS+2]; 	/* values at time (t+dt) */
float *values_d ;


/**********************************************************************
 *	Checks input values from parameters
 *********************************************************************/
void check_param(void)
{
   char tchar[20];

   /* check number of points, number of iterations */
   while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
      printf("Enter number of points along vibrating string [%d-%d]: "
           ,MINPOINTS, MAXPOINTS);
      scanf("%s", tchar);
      tpoints = atoi(tchar);
      if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
         printf("Invalid. Please enter value between %d and %d\n", 
                 MINPOINTS, MAXPOINTS);
   }
   while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
      printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
      scanf("%s", tchar);
      nsteps = atoi(tchar);
      if ((nsteps < 1) || (nsteps > MAXSTEPS))
         printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
   }

   printf("Using points = %d, steps = %d\n", tpoints, nsteps);

}

__global__ void parallel_fnc(float *values_d, int tpoints, int nsteps){
/**********************************************************************
 *     Initialize points on line
 *********************************************************************/
   int i, k;
   float x, fac, tmp;
   float value, newval, oldval ;
   fac = 2.0 * PI ;
   k = BLOCK_SIZE * blockIdx.x + threadIdx.x + 1 ;
   tmp = tpoints - 1 ;
   x = ( k - 1 ) / tmp ;
   value = sin(fac * x) ;
   oldval = value ;

/**********************************************************************
 *      Update all values along line a specified number of times
 *********************************************************************/
   if ( k <= tpoints ){
      for ( i = 1 ; i <= nsteps ; i++ ){
         if ( (k==1) || (k==tpoints) )
            newval = 0.0 ;
         else
            /*original do_math here*/
            newval = 2.0 * value - oldval - 0.09 * 2.0 * value ;
         oldval = value ;
         value = newval ;
      }
      values_d[ k ] = value ;
   }
}

/**********************************************************************
 *     Print final results
 *********************************************************************/
void printfinal()
{
   int i;

   for (i = 1; i <= tpoints; i++) {
      printf("%6.4f ", values[i]);
      if (i%10 == 0)
         printf("\n");
   }
}

/**********************************************************************
 *	Main program
 *********************************************************************/
int main(int argc, char *argv[])
{

   int size , idx ;
	sscanf(argv[1],"%d",&tpoints);
	sscanf(argv[2],"%d",&nsteps);
	check_param();

   size = (tpoints + 1) * sizeof(float) ;
   hipMalloc((void**)&values_d, size) ;

	printf("Initializing points on the line...\n");
	printf("Updating all points for all time steps...\n");

   idx = tpoints / BLOCK_SIZE + !(tpoints % BLOCK_SIZE == 0) ;
   parallel_fnc<<<idx, BLOCK_SIZE>>>(values_d, tpoints, nsteps) ;
   hipMemcpy(values, values_d, size, hipMemcpyDeviceToHost) ;
   hipFree(values_d) ;

	printf("Printing final results...\n");
	printfinal();
	printf("\nDone.\n\n");
	
	return 0;
}
